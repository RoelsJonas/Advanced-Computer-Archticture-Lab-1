
#include <hip/hip_runtime.h>
#include <iostream>

//void swap(int *pInt, int *pInt1);

__global__ void swap(int* in, int* out, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) out[size-1-i] = in[i];
}

int main() {
    int size = 100000;
    int* temp = (int*)(malloc(size * sizeof(int)));
    for(int i = 0; i < size; i++) {
        temp[i] = i;
    }

    int* input;
    hipMalloc(&input, size*sizeof(int));
    hipMemcpy( input, temp, size*sizeof(int), hipMemcpyHostToDevice);

    int* output;
    hipMalloc(&output, size*sizeof(int));

    int blocks = size/1024 + 1;
    int threads = size / blocks + 1;
    std::cout << blocks << ", " << threads << std::endl;

    swap<<<blocks, threads>>>(input, output, size);

    int* out = (int*)(malloc(size * sizeof(int)));
    hipMemcpy( out, output, size*sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Results: " << std::endl;
    for(int i = 0; i < size; i++) std::cout << temp[i] << ", ";
    std::cout << std::endl;
    for(int i = 0; i < size; i++) std::cout << out[i] << ", ";
    std::cout << std::endl;

    free(temp);
    free(out);
    return 0;
}



